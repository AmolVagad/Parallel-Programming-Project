#include "hip/hip_runtime.h"
// HOST Code to compute ICP for localization 



#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <cmath>
#include<vector>
#include<time.h>
#include<sys/time.h>
#include<ctime>
#include "dlib/optimization/optimization.h"
#include "dlib/optimization/find_optimal_parameters_abstract.h"
#include "dlib/optimization/optimization_bobyqa.h"
#include "dlib/optimization/find_optimal_parameters.h" 
#include "octree_code/octree.h"
#include "globals.h"

extern "C"

using namespace std;


__constant__ double R_constant[9];
__constant__ double range_x_d;
__constant__ double range_y_d;
__constant__ double range_z_d;
__constant__ double x_min_d;
__constant__ double y_min_d;
__constant__ double z_min_d;
__constant__ int bin_size_d;


// Function declarations
Matrix AllocateDeviceMatrix(const Matrix M);
double findTotalErrorInCloudOnDevice(const Matrix rt);
Vector AllocateDeviceVector(const Vector V);
#include "icp_kernel.cu"


// Creating variables to store the measurement and model data

	point_cloud_data measurement_data;
	point_cloud_data model_data;

	int bin_size = 4;

	double min_x =0;
	double min_y = 0;
	double min_z = 0;
	double range_x = 0;
	double range_y = 0;
	double range_z = 0;



	

///// For initial testing purposes carrying out rotation and translation operation on cuda//////////////////



// Define Octree 
Octree<std::vector<double>> octree_icp(bin_size); 

// Define the column vector 
typedef dlib::matrix<double,0,1> column_vector;

void cal_closest_points(Matrix rt);



// Function to carry out Rotation of given point on the device 

double* PerformRotationOnDevice(const Matrix R_h, const Matrix t_h, const Matrix Point_h, Matrix Rotated_Point_h)
{
	
	
	int size_T = t_h.width*t_h.height*sizeof(double);
	int size_Point = Point_h.width*Point_h.height*sizeof(double);
	// Declare the device variables 
	
	
	
	// Allocate memory on the device
	
	
	Matrix t_d = AllocateDeviceMatrix(t_h);
	Matrix Point_d = AllocateDeviceMatrix(Point_h);
	
	// Copy from host to device 
	
	
        hipMemcpy(t_d.elements,t_h.elements,size_T, hipMemcpyHostToDevice);
		
	hipMemcpy(Point_d.elements,Point_h.elements,size_Point, hipMemcpyHostToDevice);
	
	
	// Allocate device memory for result 
	Matrix Rotated_Point_d = AllocateDeviceMatrix(Rotated_Point_h);
	
	
	// Kernel Call 
	 
 // Setup the execution configuration
    	int blocks_w = R_h.width/TILE_WIDTH ;
    	int blocks_h = Point_h.height /TILE_WIDTH;
    
   
	if(t_h.width % TILE_WIDTH)
		blocks_w ++;

	if(R_h.height % TILE_WIDTH)
		blocks_h ++;
		         
	dim3 dimGrid(blocks_w, blocks_h, 1);

	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH,1);

	hipMemcpyToSymbol(HIP_SYMBOL(R_constant),R_h.elements,3 * 3*sizeof(double));

    // Launch the device computation threads!

     PerformRotationKernel<<<dimGrid,dimBlock>>>(t_d, Point_d, Rotated_Point_d);
		
	// Transfer Rotated Point from device to host
     hipMemcpy(Rotated_Point_h.elements, Rotated_Point_d.elements, size_T, hipMemcpyDeviceToHost);
       // Free device memory for all
     hipFree(t_d.elements); hipFree (Point_d.elements);hipFree (Rotated_Point_d.elements);
     
     return Rotated_Point_h.elements;	
	
}






// Function that calls transformation function and stores the transformed values 

void PerformTransformationToAllPoints(const Matrix R,const Matrix t, point_cloud_data * data, point_cloud_data * transformed_data, int skips)
{
	Matrix point, rotated_point;
	rotated_point.height =3;
	rotated_point.width = 1; 
	point.height = data->x_coord.size();
	point.width = 3; 
	point.elements = (double*)malloc(3*data->size*sizeof(double));
	rotated_point.elements = (double*)malloc(rotated_point.width*rotated_point.height*sizeof(double));
	for(int i  = 0; i < data->size; i++)
	{
		
		
		point.elements[i+0] = data->x_coord.at(i);
		point.elements[i+1] = data->y_coord.at(i);
		point.elements[i+2] = data->z_coord.at(i);
			
		transformed_data->x_coord.push_back(rotated_point.elements[0]);
		transformed_data->y_coord.push_back(rotated_point.elements[1]);
		transformed_data->z_coord.push_back(rotated_point.elements[2]);		 
	}
	
	transformed_data->size = transformed_data->x_coord.size();
	rotated_point.elements = PerformRotationOnDevice(R, t, point, rotated_point);
	
}





// The main function 

int main()
{
	
	
	


	ifstream infile1;
  	infile1.open ("icp_model.csv");
	char* pEnd;
	string x,y,z;

	
	// Reading data from the model map data csv file 

	
	 while(!infile1.eof()){
		getline(infile1,x, ',');
		getline(infile1,y, ',');
		getline(infile1,z);
		//getline(infile,index);
		model_data.x_coord.push_back(strtod(x.c_str(),&pEnd));
		model_data.y_coord.push_back(strtod(y.c_str(),&pEnd));
		model_data.z_coord.push_back(strtod(z.c_str(),&pEnd));
		measurement_data.index.push_back(-1);
		measurement_data.bin_index_x.push_back(-1);
		measurement_data.bin_index_y.push_back(-1);
		measurement_data.bin_index_z.push_back(-1);
	
	}
	

	
	//Remove the last elements
	model_data.x_coord.pop_back();
	model_data.y_coord.pop_back();
	model_data.z_coord.pop_back();
	model_data.size = model_data.size - 1;


	
	// Calculating the min and max values of x,y,z
	double max_x =  *max_element(model_data.x_coord.begin(),model_data.x_coord.end()) ;
	double max_y =  *max_element(model_data.y_coord.begin(),model_data.y_coord.end()) ;
	double max_z =  *max_element(model_data.z_coord.begin(),model_data.z_coord.end()) ;
	min_x =  *min_element(model_data.x_coord.begin(),model_data.x_coord.end()) ;
	min_y =  *min_element(model_data.y_coord.begin(),model_data.y_coord.end()) ;
	min_z =  *min_element(model_data.z_coord.begin(),model_data.z_coord.end()) ;
	
	
	//cout<<"Min x value "<<min_x<<endl;
	// Calculating the range
	range_x = max_x - min_x; 
	range_y = max_y - min_y; 
	range_z = max_z - min_z;
		
	//cout<<"Range x value "<<range_x<<endl;

	model_data.size = model_data.x_coord.size();
	//cout<<"model data value "<<model_data.size<<endl;
	
	
	// Storing the data into Octrees 
	for(int i= 0; i < model_data.size; i++)
	{
		int index_x = 0;
		int index_y = 0;
		int index_z = 0;
		
		index_x = floor(((model_data.x_coord.at(i)  - min_x)/range_x)*bin_size);
		index_y= floor(((model_data.y_coord.at(i)  - min_y)/range_y)*bin_size);
		index_z = floor(((model_data.z_coord.at(i)  - min_z)/range_z)*bin_size);
		
		// Boundary conditon 
		index_x = min(index_x, bin_size - 1);
		index_y = min(index_y, bin_size - 1);
		index_z = min(index_z, bin_size - 1);
		
		
		octree_icp(index_x, index_y, index_z).push_back(model_data.x_coord.at(i));
		octree_icp(index_x, index_y, index_z).push_back(model_data.y_coord.at(i));
		octree_icp(index_x, index_y, index_z).push_back(model_data.z_coord.at(i));
		
	}
	

	
			
		
	

	//Rotational function test
	double theta = 0.03;
	double point_x = 0.003;
	double point_y = 0.005;
	double point_z = 0.0;
	Matrix R;
	Matrix t;
	R.width = 3;
	R.height = 3;
	t.width = 1;
	t.height = 3;
	// Allocating memory to the matrices 

	R.elements = (double*)malloc(R.width*R.height*sizeof(double));
	t.elements = (double*)malloc(t.width*t.height*sizeof(double));

	R.elements[0] = cos(theta);R.elements[1]= -sin(theta); R.elements[2]= 0;
	R.elements[3] =sin(theta);  R.elements[4]=cos(theta); R.elements[5]= 0;
	R.elements[6] = 0; R.elements[7]= 0; R.elements[8]= 1;
	
	t.elements[0] = point_x;
	t.elements[1]= point_y;
	t.elements[2] = point_z;
	
	
	// Generate mesasurement datra by rorating the model data
	PerformTransformationToAllPoints(R, t, &model_data, &measurement_data,1);

	


	//Calling closest point.
	Matrix  rt;
	rt.width =  1;
	rt.height =  4;
	column_vector rt_lower(4), rt_upper(4);
	rt.elements = (double*)malloc(rt.width*rt.height*sizeof(double));
	rt.elements[0] = 0;rt.elements[1] = 0;
	rt.elements[2] = 0;rt.elements[3] = 0;
	rt_lower = -1.0, -1.0,-1.0,-1.0;
	rt_upper = 1.0, 1.0, 1.0, 1.0;

	double temp_error = 0;	
	double cpu_starttime , cpu_endtime;
	temp_error = findTotalErrorInCloudOnDevice(rt);
	cpu_starttime = clock();
	cal_closest_points(rt);
	cpu_endtime = clock();
	cout<<"The time taken for calculation of closest point = "<<((cpu_endtime - cpu_starttime)/CLOCKS_PER_SEC)<<endl;
/*
	double final_error = 0;
	// time measurement variables 


	double cpu_starttime , cpu_endtime;
	for(int i = 0; i<20; i++)
	{
		cout<<"iteration #: "<<i<<endl;
		cpu_starttime = clock();
		cal_closest_points(rt);
		cpu_endtime = clock();
		cout<<"The time taken for calculation = "<<((cpu_endtime - cpu_starttime)/CLOCKS_PER_SEC)<<endl;

		final_error = find_optimal_parameters(0.01, 0.000000001,100000, rt, rt_lower, rt_upper,findTotalErrorInCloud);
		cout<<"Rt parameters "<<rt<<endl;
		cout<<"current error: "<<final_error<<endl;
		
	}
	//cout<<"Error after optimization "<<final_error<<endl;
*/
	
	




	return 0;
}


//Calculating the closest point
void cal_closest_points(Matrix rt)
{
	point_cloud_data transformed_data;

	Matrix R_h, t_h;
	R_h.width = 3;
	R_h.height = 3;
	t_h.width = 1;
	t_h.height = 3;
	// Allocating memory to the matrices 

	R_h.elements = (double*)malloc(R_h.width*R_h.height*sizeof(double));
	t_h.elements = (double*)malloc(t_h.width*t_h.height*sizeof(double));

	R_h.elements[0] = cos(rt.elements[0]);R_h.elements[1]= -sin(rt.elements[0]); R_h.elements[2]= 0;
	R_h.elements[3] = sin(rt.elements[0]);  R_h.elements[4]= cos(rt.elements[0]); R_h.elements[5]= 0;
	R_h.elements[6] = 0; R_h.elements[7]= 0; R_h.elements[8]= 1;
	
	t_h.elements[0] = rt.elements[1];
	t_h.elements[1] = rt.elements[2];
	t_h.elements[2] = rt.elements[3];


	PerformTransformationToAllPoints(R_h, t_h, &measurement_data, &transformed_data,1);
	/*
	//Calculate the bin index of the points
	double * x_coord_dev;
	hipMalloc((void**)&x_coord_dev, transformed_data.size*sizeof(double));
	hipMemcpy(x_coord_dev, transformed_data.x_coord.data(), transformed_data.size*sizeof(double), hipMemcpyHostToDevice);

	double * y_coord_dev;
	hipMalloc((void**)&y_coord_dev, transformed_data.size*sizeof(double));
	hipMemcpy(y_coord_dev, transformed_data.y_coord.data(), transformed_data.size*sizeof(double), hipMemcpyHostToDevice);

	double * z_coord_dev;
	hipMalloc((void**)&z_coord_dev, transformed_data.size*sizeof(double));
	hipMemcpy(z_coord_dev, transformed_data.z_coord.data(), transformed_data.size*sizeof(double), hipMemcpyHostToDevice);

	int * bin_x_d;
	hipMalloc((void**)&bin_x_d, transformed_data.size*sizeof(int));

	int * bin_y_d;
	hipMalloc((void**)&bin_y_d, transformed_data.size*sizeof(int));

	int * bin_z_d;
	hipMalloc((void**)&bin_z_d, transformed_data.size*sizeof(int));

	hipMemcpyToSymbol(HIP_SYMBOL(&range_x_d), &range_x, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(&range_y_d), &range_y, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(&range_z_d), &range_z, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(&x_min_d), &min_x, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(&y_min_d), &min_y, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(&z_min_d), &min_z, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(&bin_size_d), &bin_size, sizeof(int));


	//Calculate the number of blocks and grid
	dim3 block, grid;
	block.x = TILE_WIDTH;
	block.y = 1;
	block.z = 1;
	if(transformed_data.size%block.x == 0)
		grid.x = transformed_data.size%block.x;
	else
		grid.x = transformed_data.size/block.x;
	grid.y = 1;
	grid.z = 1;

	find_bin_x_kernel<<<grid, block>>>(x_coord_dev, transformed_data.size, bin_x_d);
	find_bin_y_kernel<<<grid, block>>>(y_coord_dev, transformed_data.size, bin_y_d);
	find_bin_z_kernel<<<grid, block>>>(z_coord_dev, transformed_data.size, bin_z_d);

	hipDeviceSynchronize();

	int * bin_x = (int*)malloc(transformed_data.size*sizeof(int));
	int * bin_y = (int*)malloc(transformed_data.size*sizeof(int));
	int * bin_z = (int*)malloc(transformed_data.size*sizeof(int));

	hipMemcpy(bin_x, bin_x_d, transformed_data.size*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(bin_y, bin_y_d, transformed_data.size*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(bin_z, bin_z_d, transformed_data.size*sizeof(int), hipMemcpyDeviceToHost);
	*/

	//Calculate the closest point
	double * x_coord_model_d;
	hipMalloc((void**)&x_coord_model_d, model_data.size*sizeof(double));
	hipMemcpy(x_coord_model_d, model_data.x_coord.data(), model_data.size*sizeof(double), hipMemcpyHostToDevice);

	double * y_coord_model_d;
	hipMalloc((void**)&y_coord_model_d, model_data.size*sizeof(double));
	hipMemcpy(y_coord_model_d, model_data.y_coord.data(), model_data.size*sizeof(double), hipMemcpyHostToDevice);

	double * z_coord_model_d;
	hipMalloc((void**)&z_coord_model_d, model_data.size*sizeof(double));
	hipMemcpy(z_coord_model_d, model_data.z_coord.data(), model_data.size*sizeof(double), hipMemcpyHostToDevice);

	double * distance_d;
	hipMalloc((void**)&distance_d, model_data.size*sizeof(double));

	int * bin_index_d;	
	hipMalloc((void**)&bin_index_d, model_data.size*sizeof(int));

	
	
	for(int i=0; i < transformed_data.size; i++)
	{	
		dim3 block, grid;
		block.x = TILE_WIDTH;
		block.y = 1;
		block.z = 1;
		if(transformed_data.size%block.x == 0)
			grid.x = transformed_data.size/block.x;
		else
			grid.x = transformed_data.size/block.x + 1;
		grid.y = 1;
		grid.z = 1;
		cout<<"Check grid "<<grid.x<<endl;
		int size_data = model_data.size;
		double point_x = transformed_data.x_coord[i];
		double point_y = transformed_data.y_coord[i];
		double point_z = transformed_data.z_coord[i];
		
		find_closest_point_i<<<grid, block>>>(point_x, point_y, point_z, x_coord_model_d, y_coord_model_d, z_coord_model_d, bin_index_d + i, distance_d + i, size_data);
			
		while(grid.x > 1)
		{	
			//cout<<"Check grid 2 "<<grid.x<<endl;
			size_data = grid.x;
			if(grid.x%block.x == 0)
				grid.x = grid.x/block.x;
			else
				grid.x = grid.x/block.x + 1;
			find_closest_point_2<<<grid,block>>>(distance_d + i, bin_index_d + i, size_data);

		}		
	}
	



}









// Function to find the total error in cloud






double findTotalErrorInCloudOnDevice(const Matrix rt) //This function can be written parallelly using Atomic Add operation
{
	//iterations++;
	double icp_error = 0.0;
	point_cloud_data transformed_data;
	Matrix R, t;
        R.width = 3;R.height =3;t.height =3;t.width = 1;
	R.elements = (double*)malloc(R.width*R.height*sizeof(double));
	t.elements = (double*)malloc(t.width*t.height*sizeof(double));

	
	R.elements[0] = cos(rt.elements[0]);R.elements[1] = -sin(rt.elements[0]);R.elements[2] = 0; R.elements[3] = sin(rt.elements[0]);R.elements[4] = cos(rt.elements[0]);R.elements[5] = 0;
	R.elements[6] = 0; R.elements[7] = 0; R.elements[8] = 1;
	t.elements[0] = rt.elements[1];
	t.elements[1] =  rt.elements[2];
	t.elements[2] =  rt.elements[3];
	//cout<<"Check measurement data element "<<measurement_data.x_coord.at(0)<<endl;
	PerformTransformationToAllPoints(R, t, &measurement_data, &transformed_data,1);


	// Creating device variables 

	Vector bin_index_x_device,bin_index_y_device, bin_index_z_device,index_device,transform_x_device,transform_y_device,transform_z_device ;
	
	

	index_device.size = measurement_data.index.size();

	
	bin_index_x_device.size = measurement_data.bin_index_x.size();
	bin_index_y_device.size = measurement_data.bin_index_y.size();
	bin_index_z_device.size = measurement_data.bin_index_z.size();

	transform_x_device.size = transformed_data.x_coord.size();
	transform_y_device.size = transformed_data.y_coord.size();
	transform_z_device.size = transformed_data.z_coord.size();


	int size_var = bin_index_x_device.size;

	float* error_on_hostt,error_on_device ;
	
	// Allocate memory and copy to device 	



	hipMalloc((void**)&error_on_device,sizeof(float));
	
	
	hipMalloc((void**)&transform_x_device.elements,transform_x_device.size*sizeof(double));
	
	hipMemcpy(transform_x_device.elements,transformed_data.x_coord.data(),transform_x_device.size*sizeof(double), hipMemcpyHostToDevice);




	hipMalloc((void**)&transform_y_device.elements,transform_y_device.size*sizeof(double));
	
	hipMemcpy(transform_y_device.elements,transformed_data.y_coord.data(),transform_y_device.size*sizeof(double), hipMemcpyHostToDevice);


	hipMalloc((void**)&transform_z_device.elements,transform_z_device.size*sizeof(double));
	
	hipMemcpy(transform_z_device.elements,transformed_data.z_coord.data(),transform_z_device.size*sizeof(double), hipMemcpyHostToDevice);
		
	


	
	hipMalloc((void**)&bin_index_x_device.elements,size_var*sizeof(int));
	
	hipMemcpy(bin_index_x_device.elements,measurement_data.bin_index_x.data(),size_var*sizeof(int), hipMemcpyHostToDevice);


	hipMalloc((void**)&bin_index_y_device.elements,size_var*sizeof(int));
	
	hipMemcpy(bin_index_y_device.elements,measurement_data.bin_index_y.data(),size_var*sizeof(int), hipMemcpyHostToDevice);


	hipMalloc((void**)&bin_index_z_device.elements,size_var*sizeof(int));
	
	hipMemcpy(bin_index_z_device.elements,measurement_data.bin_index_z.data(),size_var*sizeof(int), hipMemcpyHostToDevice);



	hipMalloc((void**)&bin_index_z_device.elements,size_var*sizeof(int));
	
	hipMemcpy(bin_index_z_device.elements,measurement_data.bin_index_z.data(),size_var*sizeof(int), hipMemcpyHostToDevice);



	// Kernel Call 

//-------Add kernel call function here --------------------

	
	
	

	

	
	//cout<<"Measurement data size "<<measurement_data.size<<endl;
	/*
	



	for(int i = 0; i < measurement_data.size; i++)
	{
		
		int j = measurement_data.index[i];
		int x_Idx = measurement_data.bin_index_x[i];
		int y_Idx = measurement_data.bin_index_y[i];
		int z_Idx = measurement_data.bin_index_z[i];
		

		
		icp_error +=sqrt(pow((transformed_data.x_coord[i] - octree_icp(x_Idx, y_Idx, z_Idx)[3*j]),2) + pow((transformed_data.y_coord[i] - octree_icp(x_Idx, y_Idx, z_Idx)[3*j + 1]),2) + pow((transformed_data.z_coord[i] - octree_icp(x_Idx, y_Idx, z_Idx)[3*j + 2]),2)); 

		
	}
	
*/
	return icp_error;
}

// Function to allocate matrix memory on the device
 
Matrix AllocateDeviceMatrix(const Matrix M)
{
    Matrix Mdevice = M;
    int size = M.width * M.height * sizeof(float);
    hipMalloc((void**)&Mdevice.elements, size);
    
    return Mdevice;
}
/*
Vector AllocateDeviceVector(std::vector<int> V)
{
    std::vector<int> Vdevice = V;
    int size = V.size() * sizeof(int);
    hipMalloc((void**)&Vdevice, size);
    return Vdevice;
}

*/












	
